#include "hip/hip_runtime.h"
// adaptation of Pavel's imreconstruction code for openCV

#include "internal_shared.hpp"
#include "opencv2/gpu/device/vecmath.hpp"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>
#include <thrust/tuple.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/copy.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/count.h>


#define MAX_THREADS		256
#define XX_THREADS	4
#define XY_THREADS	32
#define NEQ(a,b)    ( (a) != (b) )

#define WARP_SIZE 32

using namespace cv::gpu;
using namespace cv::gpu::device;


namespace nscale { namespace gpu {

// 3 * WARP_SIZE.  first WARP_SIZE is dummy so to avoid warp divergence in warp_scan.  second WARP_SIZE is the scan area.  third WARP_SIZE is boolean marking the items to copy later.
template<class T> 
__device__ int warp_mark(const T* s_in_data, volatile int* s_mark, const int idx) {
	s_mark[idx - WARP_SIZE] = 0;  // extra padding for the scan part...
	s_mark[idx + WARP_SIZE] = s_mark[idx] = (s_in_data[threadIdx.x] > 0 ? 1, 0);
	return s_mark[idx];
} 


// adapted from CudPP.  exclusive should be 1 or 0.
template<int maxlevel, int exclusive>
__device__ int warp_scan(volatile int* s_mark, const int idx) {
	int t = s_mark[idx];
	if (0 <= maxlevel) { s_mark[idx] = t = t + s_mark[idx - 1]; }
	if (1 <= maxlevel) { s_mark[idx] = t = t + s_mark[idx - 2]; }
	if (2 <= maxlevel) { s_mark[idx] = t = t + s_mark[idx - 4]); }
	if (3 <= maxlevel) { s_mark[idx] = t = t + s_mark[idx - 8]); }
	if (4 <= maxlevel) { s_mark[idx] = t = t + s_mark[idx -16]); }
	return s_mark[idx - exclusive]; // exclusive  scan.
} 

// out_data points to the starting position where the compacted stuff is to go.
template<class T> 
__device__ int warp_select(const T* s_in_data, const int* s_mark, volatile T* s_out_data, const int idx) {
	const  int pos = s_mark[idx] - 1;  // new position
	if (s_mark[idx + WARP_SIZE] > 0) { s_out_data[pos] = s_in_data[threadIdx.x]; }
	return s_mark[idx];
} 

// unordered
template<class T>
__global__ void unordered_compact(const T* in_data, const int dataSize, volatile T* out_data, volatile int* queue_size) {
	// initialize the variables
	const int idx = threadIdx.x & (WARP_SIZE - 1);
	const int warpId = threadIdx.x >> 5;
	const int x = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ volatile int totals[WARP_SIZE];
	__shared__ volatile int s_mark[NUM_WARPS][WARP_SIZE * 3 + 1];
	__shared__ volatile T s_in_data[BLOCK_SIZE];
	__shared__ volatile T s_out_data[BLOCK_SIZE];
	int curr_pos;

	// copy in data
	totals[idx] = 0;
	s_out_data[threadIdx.x] = 0;
	s_in_data[threadIdx.x] = 0;
	if (x < dataSize) s_in_data[threadIdx.x] = in_data[x];
	__syncthreads();

	// compact within this block
	warp_mark(s_in_data, s_mark[warpId] + WARP_SIZE, idx);  // mark the data to be processed
	warp_scan<5, 0>(s_mark[warpId] + WARP_SIZE, idx);  // perform the in warp scan
	totals[idx] += warp_select(s_in_data, s_mark[warpId] + WARP_SIZE, s_out_data + totals[WARP_SIZE-1], idx);  // compact the data into the global space.
	__syncthreads();

	//copy the data back out.  this block will get a place to write using atomic add.  resulting queue has the blocks shuffled
	if (totals[WARP_SIZE-1] > 0) {
		if (threadIdx.x == 0) curr_pos = atomicAdd(queue_size, totals[WARP_SIZE-1]); // only done by first thread in the block
		if (threadIdx.x < totals[WARP_SIZE-1]) out_data[curr_pos + threadIdx.x] = s_out_data[threadIdx.x];   // dont need to worry about dataSize.  queue size is smaller...
	}
}

// gapped.  so need to have anothr stop to copy stuff over...  block_pos stores the lengths of the blcok queue for each block
template<class T>
__global__ void gapped_compact(const T* in_data, const int dataSize, volatile T* out_data, volatile int* block_pos) {
	// initialize the variables
	const int idx = threadIdx.x & (WARP_SIZE - 1);
	const int warpId = threadIdx.x >> 5;
	const int x = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ volatile int totals[WARP_SIZE];
	__shared__ volatile int s_mark[NUM_WARPS][WARP_SIZE * 3 + 1];
	__shared__ volatile T s_in_data[BLOCK_SIZE];
	__shared__ volatile T s_out_data[BLOCK_SIZE];

	// copy in data
	totals[idx] = 0;
	s_out_data[threadIdx.x] = 0;
	s_in_data[threadIdx.x] = 0;
	if (x < dataSize) s_in_data[threadIdx.x] = in_data[x];
	__syncthreads();

	// compact within this block
	warp_mark(s_in_data, s_mark[warpId] + WARP_SIZE, idx);  // mark the data to be processed
	warp_scan<5,0>(s_mark[warpId] + WARP_SIZE, idx);  // perform the in warp scan
	totals[idx] += warp_select(s_in_data, s_mark[warpId] + WARP_SIZE, s_out_data + totals[WARP_SIZE-1], idx);  // compact the data into the global space.
	__syncthreads();

	//copy the data back out.  this block will get a place to write using atomic add.  resulting queue has the blocks shuffled
	if (x < dataSize) out_data[x] = s_out_data[threadIdx.x];
	if (threadIdx.x == 0) block_pos[blockIdx.x] = totals[WARP_SIZE - 1];
}

//1024 threads - warpscan all, then 1 warp to scan, then everyone add.
__device__ void block_scan(volatile int* data, volatile int * s_mark) {
	const int idx = threadIdx.x & (WARP_SIZE - 1);
	const int warpId = threadIdx.x >> 5;

	// initialize data:
	s_mark[NUM_WARPS][warpId] = 0;

	s_mark[warpId][idx+WARP_SIZE] = data[threadIdx.x];


	int out = warp_scan<5, 0>(s_mark[warpId] + WARP_SIZE, idx);

	if (idx == WARP_SIZE - 1) s_mark[NUM_WARPS][warpId + WARP_SIZE] = out;
	__syncthreads();

	// do the second pass
	if (threadIdx.x < WARP_SIZE) {
		warp_scan<5, 1>(s_mark[NUM_WARPS] + WARP_SIZE, idx);  // exclusive scan
	}
	__synthreads();

	// add back
	data[threadIdx.x] = s_mark[warpId][idx + WARP_SIZE] + s_mark[NUM_WARPS][warpId + WARP_SIZE - 1];   // using exclusive scan results.
}

__global__ void scan() {

	__shared__ volatile int s_mark[NUM_WARPS + 1][WARP_SIZE * 3 + 1];


	// now do the multiple block version...
}


template<class T>
__global__ void clear(volatile T* out_data, const int dataSize) {
	const int x = threadIdx.x + blockDim.x * blockIdx.x;
	if (x < dataSize) out_data[x] = 0;
}

// step 2 of the compacting.  assumes that within each block the values have already been compacted.
template<class T>
__global__ void compact(const T* in_data, const int* block_pos, volatile T* out_data ) {
	const int x = threadIdx.x + blockDim.x * blockIdx.x;
	const int pos = block_pos[blockIdx.x];
	const int len = block_pos[blockIdx.x + gridDim.x];

	if (threadIdx.x < len) out_data[pos + threadIdx.x] = in_data[threadIdx.x];
}
/*
template<typename T, typename TN>
struct InitialImageToQueue : public thrust::unary_function<TN, int>
{
    __host__ __device__
        int operator()(const TN& pixel) const
        {
		T center = thrust::get<1>(pixel);
		T curr;
		int id = thrust::get<0>(pixel);
		curr = thrust::get<2>(pixel);
		if (curr < center && curr < thrust::get<6>(pixel)) return id;
		curr = thrust::get<3>(pixel);
		if (curr < center && curr < thrust::get<7>(pixel)) return id;
		curr = thrust::get<4>(pixel);
		if (curr < center && curr < thrust::get<8>(pixel)) return id;
		curr = thrust::get<5>(pixel);
		if (curr < center && curr < thrust::get<9>(pixel)) return id;
		return -1;
        }
};
*/





// connectivity:  need to have border of 0 ,and should be continuous
template <typename T>
unsigned int SelectTesting(const T* in_data, volatile T* out_data, hipStream_t stream) {

	dim3 threadsx( XX_THREADS, XY_THREADS );
	dim3 blocksx( divUp(sy, threadsx.y) );
	dim3 threadsy( MAX_THREADS );
	dim3 blocksy( divUp(sx, threadsy.x) );

	// stability detection



	typedef typename thrust::device_ptr<T> PixelIterator;

//		typedef typename thrust::tuple<int, T, T, T, T, T> PixelNeighborhood;
//		typedef typename thrust::tuple<thrust::counting_iterator<int>, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator> WindowedImage;
//		typedef typename thrust::zip_iterator<WindowedImage> WindowedPixelIterator;

	typedef typename thrust::tuple<signed int, T, T, T, T, T, T, T, T, T> ReconNeighborhood;
	typedef typename thrust::tuple<signed int, T, T, T> ReconNeighborhood2;
	typedef typename thrust::tuple<thrust::counting_iterator<int>, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator, PixelIterator> ReconImage;
	typedef typename thrust::zip_iterator<ReconImage> ReconPixelIterator;

	typedef typename thrust::device_vector<int> Queue;
	typedef typename Queue::iterator QueueIterator;
	typedef typename thrust::tuple<int, int, int, int> QueueElement;


	thrust::counting_iterator<int> ids;
//		WindowedImage markerImg = thrust::make_tuple(ids, q_ym1xm1, q_ym1, q_ym1xp1, q_xm1, q, q_xp1, q_yp1xm1, q_yp1, q_yp1xp1);
//		WindowedImage markerImgEnd = thrust::make_tuple(ids+area, q_ym1xm1+area, q_ym1+area, q_ym1xp1+area, q_xm1+area, q+area, q_xp1+area, q_yp1xm1+area, q_yp1+area, q_yp1xp1+area);
//		WindowedImage maskImg = thrust::make_tuple(ids, p_ym1xm1, p_ym1, p_ym1xp1, p_xm1, p, p_xp1, p_yp1xm1, p_yp1, p_yp1xp1);
//		ReconPixelIterator mask_last = thrust::make_zip_iterator(thrust::make_tuple(p_ym1xm1+area, p_ym1+area, p_ym1xp1+area, p_xm1+area, p+area, p_xp1+area, p_yp1xm1+area, p_yp1+area, p_yp1xp1+area));

	ReconImage markermaskNp = thrust::make_tuple(ids, q, q_xp1, q_yp1xm1, q_yp1, q_yp1xp1, p_xp1, p_yp1xm1, p_yp1, p_yp1xp1);
	ReconImage markermaskNpEnd = thrust::make_tuple(ids+area, q+area, q_xp1+area, q_yp1xm1+area, q_yp1+area, q_yp1xp1+area, p_xp1+area, p_yp1xm1+area, p_yp1+area, p_yp1xp1+area);
	ReconPixelIterator image_first = thrust::make_zip_iterator(markermaskNp);
	ReconPixelIterator image_last = thrust::make_zip_iterator(markermaskNpEnd); 

	// put the candidates into the queue
	int queueSize = area;
	Queue sparseQueue(queueSize, -1);

	// can change into transform_iterator to use in the copy operation.  the only challenge is don't know queue size, and would still need to compact later...
	// mark
	thrust::transform(image_first, image_last, sparseQueue.begin(), InitialImageToQueue<T, ReconNeighborhood>());
	// select
	queueSize = thrust::count_if(sparseQueue.begin(), sparseQueue.end(), GreaterThanConst<int>(-1));

	Queue testQueue(area, -1);

	// compact the queue
	Queue denseQueue(queueSize, 0);
	QueueIterator denseQueue_end = thrust::copy_if(sparseQueue.begin(), sparseQueue.end(), denseQueue.begin(), GreaterThanConst<int>(-1));
	QueueIterator sparseQueue_end;

	thrust::device_vector<bool> dummy(area, false);
	printf("number of entries in sparseQueue: %d, denseQueue: %d \n", queueSize, denseQueue_end - denseQueue.begin());
	int iterations = 0;
	int total = 0;
	while (queueSize > 0 && iterations < 10000) {
		++iterations;
		total += queueSize;

//			printf("here\n");
		// allocate some memory
//			sparseQueue.resize(queueSize * 8);  // 8 neighbors
//			thrust::fill(sparseQueue.begin(), sparseQueue.end(), -1);
		// also set up as 8 devPtrs
//			QueueIterator ym1xm1 = sparseQueue.begin();
//			QueueIterator ym1 = ym1xm1+queueSize;
//			QueueIterator ym1xp1 = ym1+queueSize;
//			QueueIterator xm1 = ym1xp1+queueSize;
//			QueueIterator xp1 = xm1+queueSize;
//			QueueIterator yp1xm1 = xp1+queueSize;
//			QueueIterator yp1 = yp1xm1+queueSize;
//			QueueIterator yp1xp1 = yp1+queueSize;
//						printf("here3\n");
//			dummy.resize(queueSize);

		// sort the queue by the value
		sparseQueue_end = thrust::copy(denseQueue.begin(), denseQueue.end(), sparseQueue.begin());
		thrust::stable_sort_by_key(thrust::make_permutation_iterator(q, sparseQueue.begin()),
				thrust::make_permutation_iterator(q, sparseQueue_end),
				denseQueue.begin());

		thrust::fill(dummy.begin(), dummy.end(), false);
		thrust::for_each(denseQueue.begin(), denseQueue.end(), Propagate<T>(thrust::raw_pointer_cast(q),
				thrust::raw_pointer_cast(p), thrust::raw_pointer_cast(&*dummy.begin()), sx));


		// and prepare the queue for the next iterations.
			//sparseQueue_end = thrust::unique(sparseQueue.begin(), sparseQueue.end());
			queueSize = thrust::count_if(dummy.begin(), dummy.end(), thrust::identity<bool>());
//			printf("here 7 : queueSize =%d \n", queueSize);

		denseQueue.resize(queueSize);
		thrust::fill(denseQueue.begin(), denseQueue.end(), -1);

		denseQueue_end = thrust::copy_if(ids, ids+area, dummy.begin(), denseQueue.begin(), thrust::identity<bool>());
		printf("number of entries in queue: %d \n", denseQueue_end - denseQueue.begin());

	}


	if (stream == 0) cudaSafeCall(hipDeviceSynchronize());
	else cudaSafeCall( hipStreamSynchronize(stream));
	cudaSafeCall( hipGetLastError());

	printf("iterations: %d, total: %d\n", iterations, total);
	return total;

}

template unsigned int imreconQueueIntCaller<unsigned char>(unsigned char*, unsigned char*, const int, const int,
	const int, hipStream_t );
}}
