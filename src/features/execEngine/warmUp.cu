#include "cutil.h"
#include <string.h>
#include <stdio.h>

void warmUp(int device){
	void *d_data;
	int *h_data = (int *) malloc(sizeof(int));
	h_data[0] = 10;

	// choose the appropriate device
	hipSetDevice(device);

	hipMalloc( (void**)&d_data, sizeof(int) );

	hipMemcpy(d_data, h_data, sizeof(int), hipMemcpyHostToDevice );

	hipFree(d_data);
	free(h_data);
}
