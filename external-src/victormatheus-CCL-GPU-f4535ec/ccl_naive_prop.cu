#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>  

#include "textures.cuh"

namespace naive_prop {

const int BLOCK_X = 16;
const int BLOCK_Y = 16;

__global__ void PROP_prescan(int* R, int w, int h) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int index = x+y*w;

    if (x < w && y < h) {
        R[index] = index;
    }
}

__global__ void PROP_scan(int* R, int w, int h, int* d_stop) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int index = x+y*w;

    if (x < w && y < h) {
        unsigned char v = tex2D(imgtex, x, y);
        int label = R[index];
        int newlabel = w*h;
 
        if (y>0 && tex2D(imgtex, x, y-1) == v) {
            newlabel = min(newlabel, R[index-w]);
        }
        if (y<h-1 && tex2D(imgtex, x, y+1) == v) {
            newlabel = min(newlabel, R[index+w]);
        }
        if (x>0 && tex2D(imgtex, x-1, y) == v) {
            newlabel = min(newlabel, R[index-1]);
        }
        if (x<w-1 && tex2D(imgtex, x+1, y) == v) {
            newlabel = min(newlabel, R[index+1]);
        }

        if (newlabel< label) {
            R[index] = newlabel;
            *d_stop = 0;
        }
    }
}

void CCL(unsigned char* img, int w, int h, int* label) {
    hipError_t err;

    hipArray* imgarray;
    hipChannelFormatDesc uchardesc = 
        hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&imgarray, &uchardesc, w, h);

    int* R;
    hipMalloc((void**)&R, w*h*sizeof(int));

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("startERROR: %s\n", hipGetErrorString(err));
        return;
    }

    hipChannelFormatDesc intdesc = 
        hipCreateChannelDesc<int>();
    hipBindTextureToArray(imgtex, imgarray, uchardesc);
    hipBindTexture(NULL, Rtex, R, intdesc, w*h*sizeof(int));

    int stop;
    int* d_stop;
    hipMalloc((void**)&d_stop, sizeof(int));

    dim3 block (BLOCK_X, BLOCK_Y);
    dim3 grid ((w+BLOCK_X-1)/BLOCK_X,
               (h+BLOCK_Y-1)/BLOCK_Y);

    hipMemcpyToArray(imgarray, 0, 0, img, 
            w*h*sizeof(unsigned char),
            hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("midERROR: %s\n", hipGetErrorString(err));
        return;
    }

    PROP_prescan <<<grid, block>>>
        (R, w, h);

    stop = 0;
    while (stop == 0) {

        hipMemset(d_stop, 0xFF, sizeof(int));

        PROP_scan <<<grid, block>>>
            (R, w, h, d_stop);

        hipMemcpy(&stop, d_stop, sizeof(int),
                hipMemcpyDeviceToHost);
    }

    hipMemcpy(label, R, w*h*sizeof(int),
            hipMemcpyDeviceToHost); 

    hipFree(d_stop);
    hipFree(R);
    hipFreeArray(imgarray);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("endERROR: %s\n", hipGetErrorString(err));
        return;
    }
}

}
