#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>  

#define BLOCK_X 256

__global__ void
coalesced_read_1(float* f, int N) {
   int x = blockIdx.x*blockDim.x + threadIdx.x;
    
   float v;
   for (int i=0; i<100; i++) {
      v= f[x];
    }

   v *= v;
   f[x] = v;
}


__global__ void
coalesced_read_2(float* f, int N) {
   //int x = (gridDim.x-blockIdx.x-1)*blockDim.x + threadIdx.x;
   int x = blockIdx.x*blockDim.x + threadIdx.x;

   float v;

   for (int i=0; i<100; i++) {
      v = f[(x+1)%N];
    }

   v *= v;
   f[x] = v;
}


int main(int argc, char *argv[]) {

    int N = 1<<20;
    float* data;
    hipMalloc(&data, sizeof(float)*N);

    coalesced_read_1 <<<N/256, 256>>> (data, N);
    hipDeviceSynchronize();

    coalesced_read_2 <<<N/256, 256>>> (data, N);
    hipDeviceSynchronize();

    hipFree(data);

    return 0;
}
