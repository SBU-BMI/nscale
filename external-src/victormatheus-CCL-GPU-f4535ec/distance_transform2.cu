#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>  

#include "pgm.h"

#define BLOCK_SIZE 128

__device__ float
distance(float x, float y, float ex, float ey) {
    return sqrtf(powf(ex-x,2.0f)+powf(ey-y,2.0f));
}

__global__ void
euclidian_distance_transform(uchar4* img, 
        float4* dist, int w, int h) {    

    //each thread process 4 pixels
   __shared__ uchar4 img_line [BLOCK_SIZE];

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    float N = w*h;

    int4 ox = {(4*i) % w, (4*i+1) % w, (4*i+2) % w, (4*i+3) % w};
    int4 oy = {(4*i) / w, (4*i+1) / w, (4*i+2) / w, (4*i+3) / w};

    if (4*i < N) {
        float4 d = {N, N, N, N};
    
        for(int bi=0; bi<gridDim.x; bi++) {
            int nbi = (blockIdx.x+bi) % gridDim.x;
            int target = nbi*blockDim.x+threadIdx.x;

            //32-bit coalesced transaction
            img_line[threadIdx.x] = img[target];
            __syncthreads();

            for (int k=0;k<blockDim.x;k++) {
                float4 nd;
                //let's back to the original reference system
                int4 dx = {(4*target) % w, (4*target+1) % w, (4*target+2) % w, (4*target+3) % w};
                int4 dy = {(4*target) / w, (4*target+1) / w, (4*target+2) / w, (4*target+3) / w};

                //pixel 1 - x
                //pixel 2 - y

                nd.x = distance(float(ox.x),float(oy.x), float(dx.x),float(dy.x)); 
                nd.y = distance(float(ox.y),float(oy.y), float(dx.y),float(dy.y)); 
                nd.z = distance(float(ox.z),float(oy.z), float(dx.z),float(dy.z)); 
                nd.w = distance(float(ox.w),float(oy.w), float(dx.w),float(dy.w)); 

                uchar4 v = img_line[threadIdx.x];

                if (v.x == 1 && nd.x < d.x) d.x = nd.x;
                if (v.y == 1 && nd.y < d.y) d.y = nd.y;
                if (v.z == 1 && nd.z < d.z) d.z = nd.z;
                if (v.w == 1 && nd.w < d.w) d.w = nd.w;

            }
        }
    
        //128-bit coalesced transaction
        dist[i] = d;

    }
}

int main(int argc, char* argv[]) {
    hipEvent_t start,stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    //float et;
    hipError_t err;

    int w,h;
    unsigned char* img;
    img = load_ppm(argv[1], &w, &h);
    printf("%d %d\n", w, h);
    unsigned char *d_img;
    hipMalloc((void**) &d_img, w*h*sizeof(unsigned char));
    hipMemcpy(d_img, img, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
        return 1;
    }

    fprintf(stderr, "GO!");

    //-------------------
 
    float* dist = (float*)malloc(w*h*sizeof(float));
    float* d_dist;
    hipMalloc((void**) &d_dist, w*h*sizeof(float));

    dim3 block (BLOCK_SIZE,1);
    dim3 grid ((w*h+(4*BLOCK_SIZE-1))/(4*BLOCK_SIZE));
 
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
        return 1;
    }

    fprintf(stderr, "GO!");

    euclidian_distance_transform
        <<<grid, block>>> ((uchar4*)d_img, (float4*)d_dist, w, h);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
        return 1;
    }

    fprintf(stderr, "GO!");

    hipMemcpy(dist, d_dist, w*h*sizeof(float), hipMemcpyDeviceToHost);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipFree(d_img);
    hipFree(d_dist);
    free(dist);
    free(img);

    return 0;
}
